#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "IUnityInterface.h"
#include "DebugDLL.h"
#include <hipfft/hipfft.h>
#include <math_functions.h>
#include <stdio.h>

inline static bool debugError(hipError_t error, const char* msg) {
    if (error != hipSuccess) {
        DebugDLL::ss << msg << " : " << error;
        DebugDLL::log(&DebugDLL::ss.str(), Color::Red);
        return true;
    }
    return false;
}

inline static bool debugResult(hipfftResult result, const char* msg) {
    if (result != hipSuccess) {
        DebugDLL::ss << msg << " : " << result;
        DebugDLL::log(&DebugDLL::ss.str(), Color::Red);
        return true;
    }
    return false;
}


extern "C"
{
    UNITY_INTERFACE_EXPORT int cudaTest(float* data, int size)
    {
        DebugDLL::clear();

        if (data == nullptr) {
            return -1;
        }

        hipfftHandle plan;
        hipfftComplex* complexHostData;
        hipfftComplex* complexDeviceData;
        
        hipfftResult result;
        hipError_t error;

        const int byteSize = sizeof(hipfftComplex) * size;

        // init host data
        complexHostData = (hipfftComplex*)malloc(byteSize);
        for (int i = 0; i < size; i++)
            complexHostData[i] = make_hipFloatComplex(data[i], 0);


        // create device data
        error = hipMalloc((void**)&complexDeviceData, byteSize);
        if (debugError(error, "Unable to hipMalloc complexData")) {
            goto CUDA_MALLOC_ERROR;
        }


        result = hipfftPlan1d(&plan, size, HIPFFT_C2C, 1);
        if (debugResult(result, "hipfftPlan1d Failed")) {
            goto CUFFT_PLAN_ERROR;
        }

        error = hipMemcpy((void*)complexDeviceData, complexHostData, byteSize, hipMemcpyHostToDevice);
        if (debugError(error, "hipMemcpy Host => Device failed")) {
            goto CPY_TO_DEVICE_ERR;
        }

        result = hipfftExecC2C(plan, complexDeviceData, complexDeviceData, HIPFFT_FORWARD);
        if (debugResult(result, "hipfftExecC2R failed")) {
            goto CUFFT_EXEC_ERR;
        }

        error = hipDeviceSynchronize();
        if (debugError(error, "hipDeviceSynchronize failed")) {
            goto DEVICE_SYNCH_ERR;
        }

        error = hipMemcpy((void*)complexHostData, complexDeviceData, byteSize, hipMemcpyDeviceToHost);
        if (debugError(error, "hipMemcpy Device => Host failed")) {
            goto CPY_TO_HOST_ERR;
        }

        
        // free plan data
        result = hipfftDestroy(plan);
        if (debugResult(result, "hipfftDestroy failed")) {
            return -7;
        }

        error = hipFree(complexDeviceData);
        if (debugError(error, "hipFree failed")) {
            return -8;
        }

        for (int i = 0; i < size; i++) {
            auto complexNumber = complexHostData[i];
            data[i] = sqrt(complexNumber.x * complexNumber.x + complexNumber.y * complexNumber.y);
        }

        free(complexHostData);

        return 0;



    CUDA_MALLOC_ERROR:
        free(complexHostData);
        return -1;
    CUFFT_PLAN_ERROR:
        error = hipFree(complexDeviceData);
        if (debugError(error, "hipFree failed")) {
            return -8;
        }
        free(complexHostData);
        return -2;
    CPY_TO_DEVICE_ERR:
        result = hipfftDestroy(plan);
        if (debugResult(result, "hipfftDestroy failed")) {
            return -7;
        }
        free(complexHostData);
        return -3;
    CUFFT_EXEC_ERR:
        result = hipfftDestroy(plan);
        if (debugResult(result, "hipfftDestroy failed")) {
            return -7;
        }
        free(complexHostData);
        return -4;
    DEVICE_SYNCH_ERR:
        result = hipfftDestroy(plan);
        if (debugResult(result, "hipfftDestroy failed")) {
            return -7;
        }
        free(complexHostData);
        return -5;
    CPY_TO_HOST_ERR:
        result = hipfftDestroy(plan);
        if (debugResult(result, "hipfftDestroy failed")) {
            return -7;
        }
        free(complexHostData);
        return -6;
    }
}

/*
__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

// Helper function for using CUDA to add vectors in parallel
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    dim3 gridDimensions(2);
    dim3 blockDimensions(size / gridDimensions.x);
    addKernel<<<gridDimensions, blockDimensions>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

std::unique_ptr<int[]> getAArray(const unsigned int size) {
    auto a = std::unique_ptr<int[]>(new int[size]);
    for (int i = 0, int n = 1; i < size; i++, n++) {
        a[i] = n * n;
    }
    return a;
}

std::unique_ptr<int[]> getBArray(const unsigned int size) {
    auto a = std::unique_ptr<int[]>(new int[size]);
    for (int i = 0, int n = 1; i < size; i++, n++) {
        a[i] = n;
    }
    return a;
}
*/
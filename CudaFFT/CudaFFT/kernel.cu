#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "IUnityInterface.h"
#include "DebugDLL.h"
#include <hipfft/hipfft.h>
#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
std::unique_ptr<int[]> getAArray(const unsigned int size);
std::unique_ptr<int[]> getBArray(const unsigned int size);


__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

inline static bool debugError(hipError_t error, const char* msg) {
    if (error != hipSuccess) {
        DebugDLL::ss << msg << " : " << error;
        DebugDLL::log(&DebugDLL::ss.str(), Color::Red);
        return true;
    }
    return false;
}

inline static bool debugResult(hipfftResult result, const char* msg) {
    if (result != hipSuccess) {
        DebugDLL::ss << msg << " : " << result;
        DebugDLL::log(&DebugDLL::ss.str(), Color::Red);
        return true;
    }
    return false;
}


extern "C"
{
    UNITY_INTERFACE_EXPORT int cudaTest(float* data, int size)
    {
        DebugDLL::clear();

        const unsigned int arraySize = 1024;

        hipfftHandle plan;
        hipfftComplex* complexHostData;
        hipfftComplex* complexDeviceData;
        
        hipfftResult result;
        hipError_t error;

        // init host data
        complexHostData = (hipfftComplex*)malloc(sizeof(hipfftComplex) * size);
        for (int i = 0; i < size; i++)
            complexHostData[i] = make_hipFloatComplex(data[i], 0);


        // create device data
        error = hipMalloc((void**)&complexDeviceData, sizeof(hipfftComplex) * size);
        if (debugError(error, "Unable to hipMalloc complexData")) {
            goto CUDA_MALLOC_ERROR;
        }

        result = hipfftPlan1d(&plan, size, HIPFFT_C2C, 1);
        if (debugResult(result, "hipfftPlan1d Failed")) {
            goto CUFFT_PLAN_ERROR;
        }

        error = hipMemcpy((void*)complexDeviceData, complexHostData, sizeof(hipfftComplex) * size, hipMemcpyHostToDevice);
        if (debugError(error, "hipMemcpy Host => Device failed")) {
            goto CPY_TO_DEVICE_ERR;
        }

        hipfftResult result = hipfftExecC2C(plan, complexDeviceData, complexDeviceData, HIPFFT_FORWARD);
        if (debugResult(result, "hipfftExecC2R failed")) {
            goto CUFFT_EXEC_ERR;
        }

        error = hipDeviceSynchronize();
        if (debugError(error, "hipDeviceSynchronize failed")) {
            goto DEVICE_SYNCH_ERR;
        }

        error = hipMemcpy((void*)complexHostData, complexDeviceData, sizeof(hipfftComplex) * size, hipMemcpyDeviceToHost);
        if (debugError(error, "hipMemcpy Device => Host failed")) {
            goto CPY_TO_HOST_ERR;
        }


        DebugDLL::ss << "Device Copy Success! Value[777777]: {" << complexHostData[777777].x << "," << complexHostData[777777].y << "}";
        DebugDLL::log(&DebugDLL::ss.str(), Color::Blue);

        
        // ...
        result = hipfftDestroy(plan);
        if (debugResult(result, "hipfftDestroy failed")) {
            return -7;
        }

        error = hipFree(complexDeviceData);
        if (debugError(error, "hipFree failed")) {
            return -8;
        }

        free(complexHostData);

        /*
        auto a = getAArray(arraySize);
        auto b = getBArray(arraySize);

        int c[arraySize] = {0};

       
        // Add vectors in parallel
        hipError_t cudaStatus = addWithCuda(c, a.get(), b.get(), arraySize);

        if (cudaStatus != hipSuccess) {
            DebugDLL::ss << "addWithCuda failed!";
            DebugDLL::log(&DebugDLL::ss.str(), Color::Red);
            return -4;
        }

       
        DebugDLL::ss << "data[1000000]: " << data[1000000];
        DebugDLL::log(&DebugDLL::ss.str(), Color::Blue);
        */

        // hipDeviceReset must be called before exiting in order for profiling and
        // tracing tools such as Nsight and Visual Profiler to show complete traces.
        //hipError_t cudaStatus = hipDeviceReset();
        //if (cudaStatus != hipSuccess) {
        //    DebugDLL::ss << "hipDeviceReset failed!";
        //    DebugDLL::log(&DebugDLL::ss.str(), Color::Red);
        //    return -5;
        //}


        return 0;

    CUDA_MALLOC_ERROR:
        free(complexHostData);
        return -1;
    CUFFT_PLAN_ERROR:
        error = hipFree(complexDeviceData);
        if (debugError(error, "hipFree failed")) {
            return -8;
        }
        free(complexHostData);
        return -2;
    CPY_TO_DEVICE_ERR:
        result = hipfftDestroy(plan);
        if (debugResult(result, "hipfftDestroy failed")) {
            return -7;
        }
        free(complexHostData);
        return -3;
    CUFFT_EXEC_ERR:
        result = hipfftDestroy(plan);
        if (debugResult(result, "hipfftDestroy failed")) {
            return -7;
        }
        free(complexHostData);
        return -4;
    DEVICE_SYNCH_ERR:
        result = hipfftDestroy(plan);
        if (debugResult(result, "hipfftDestroy failed")) {
            return -7;
        }
        free(complexHostData);
        return -5;
    CPY_TO_HOST_ERR:
        result = hipfftDestroy(plan);
        if (debugResult(result, "hipfftDestroy failed")) {
            return -7;
        }
        free(complexHostData);
        return -6;
    }
}


// Helper function for using CUDA to add vectors in parallel
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    dim3 gridDimensions(2);
    dim3 blockDimensions(size / gridDimensions.x);
    addKernel<<<gridDimensions, blockDimensions>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

std::unique_ptr<int[]> getAArray(const unsigned int size) {
    auto a = std::unique_ptr<int[]>(new int[size]);
    for (int i = 0, int n = 1; i < size; i++, n++) {
        a[i] = n * n;
    }
    return a;
}

std::unique_ptr<int[]> getBArray(const unsigned int size) {
    auto a = std::unique_ptr<int[]>(new int[size]);
    for (int i = 0, int n = 1; i < size; i++, n++) {
        a[i] = n;
    }
    return a;
}